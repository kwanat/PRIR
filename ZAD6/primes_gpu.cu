#include "hip/hip_runtime.h"
# include <iostream>
# include <fstream>
# include <cstdlib>
# include <cmath>
# include <vector>
# include "mpi.h"

using namespace std;



struct number{		//struktura wykorzystywana w wektorze danych - zawiera informacje o wartosci liczby oraz o tym czy jest pierwsza
   unsigned long int value;
    bool prime;
};

__global__ void primeTesting (vector<number>* tab, uint sqr, uint d)         //Funkcja testująca pierwszoć liczb, przyjmuje jako argumenty zbior do tesowania, pierwiastek do ktorego testujemy oraz wielkosc zbioru testowanego
{
uint tid=blockIdx.x;                                                                          //Funkcja zwraca przetestowany zbior
uint i,j;
for (i=2;i<=sqr;i++) {			//petla zrownoleglana - kolejne liczby od 2 do pierwiastka kwadratowego z najwiekszego elementu zbioru wczytywanego
        for (j = tid; j <d; j+=blockDim.x) {		//petla wewnetrzna sprawdzajaca czy kolejne liczby wektora tab dziela sie przez aktualna wartosc zmiennej i
                if((tab[j].value%i==0)&(tab[j].value!=i)) //jesli tak liczba uznawana jest za zlozona, dodatkowo sprawdzamy czy liczba nie jest rowna obecnemu dzielnikowi (zasada pierwszosci)
                    tab[j].prime=false;
        }
    }
}

int main(int argc, char** argv) {
    int blockNumber=1;
    
    ifstream file;  	//plik wejsciowy
    unsigned int maxval=0;  //zmienna przechowująca wartosc maksymalna z testowanego pliku
    number fromfile;	      // pojedyncza liczba z pliku wraz z informacja o pierwszosci
    
    hipEvent_t start, stop; //deklaracja zmiennych licznika
    float elapsedTime; 
    hipError_t error;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    

    if (argc != 2) {				//sprawdzenie ilosci argumentow podanych przy wywolaniu programu
        cout << "The number of arguments is invalid"<<endl;
        exit(1);
    }
    file.open(argv[1]);
	if (file.fail()){			//Sprawdzenie poprawnosci otwartego pliku
		cout<<"Could not open file to read."<<endl;
		exit(1);
	}

    vector<number> tab;				//utworzenie wektora liczb

    while (file >> fromfile.value) {		//pobranie danych z pliku do wektora tab
        fromfile.prime=true;			//domniemanie pierwszosci liczby
        tab.push_back(fromfile);		//zapisanie liczby w wektorze tab
        if(fromfile.value>maxval)		//poszukiwanie liczby maksymalnej ze zbioru
            maxval=fromfile.value;

    }

   				
    uint sqr=sqrt(maxval);			//pierwiastek z liczby maksymalnej
    uint d=tab.size();          //zmienna pomocnicza rozmiar wektora danych

    vector<number>* tab2;
    error=hipMalloc( (void**)&tab2, d * sizeof(number) );
    error = hipMemcpy(tab2, tab, d * sizeof(number), hipMemcpyHostToDevice);
    hipEventRecord(start);

    primeTesting <<< blockNumber, 1 >>> (tab2, sqr,d);

    hipEventRecord(stop); //zatrzymanie licznika i zczytanie czasu obliczen
    error = hipEventSynchronize(stop);
    vector<number>* reult= new vector<number>();
    result->reserve( d );
    error = hipMemcpy(result, tab2, d * sizeof(number), hipMemcpyDeviceToHost);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Czas : %f ms\n", elapsedTime);
    for (uint i=0;i<(* result).size();i++)				//wypisanie liczb z  wektora tab wraz z informacją czy są pierwsze
        if((* result)[i].prime==true)
            cout<<(* result)[i].value<<": prime"<<endl;
        else
            cout<<(* result)[i].value<<": composite"<<endl;
	                  
    return 0;                  

}
