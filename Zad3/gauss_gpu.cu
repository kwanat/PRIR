#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <cstdlib>
#include <vector>
#include <cmath>

#include<opencv2/imgproc/imgproc.hpp>
#include<opencv2/core/core.hpp>
#include<opencv2/highgui/highgui.hpp>
 
using namespace cv;
using namespace std;
 
__global__ void gaussianBlur(unsigned char* R, unsigned char* G, unsigned char* B,
                             unsigned char* resultRed, unsigned char* resultGreen, unsigned char* resultBlue,
                             int blockSize, int threadSize, int rowsNumber, int columnsNumber) {
    
    //Initialize Data
const int mask[5][5] = {            //druga wersja filtru dolnoprzepustowego wykorzystującego funkcję Gaussa
        {1, 1, 2, 1, 1},
        {1, 2, 4, 2, 1},
        {2, 4, 8, 4, 2},
        {1, 2, 4, 2, 1},
        {1, 1, 2, 1, 1}
    };
    int SumMask = 52;   //suma liczb w filtrze
    unsigned int row, column, x, y, red, green, blue;
    int  startForBlock, startForThread;
    //Calculate start position for Block:
    if (blockIdx.x < (rowsNumber + 4) % gridDim.x) {
        startForBlock = blockIdx.x * blockSize + blockIdx.x;
        blockSize++;
    }
    else {
        startForBlock = blockIdx.x * blockSize + (rowsNumber+4) % gridDim.x;
    }
    //Calcualte start position for Thread
    if (threadIdx.x < (columnsNumber + 4) % blockDim.x) {
        startForThread = threadIdx.x*threadSize + threadIdx.x;
        threadSize++;
    }
    else {
        startForThread = threadIdx.x*threadSize + (columnsNumber+4) % blockDim.x;
    }
    //Do Calculations
    for (row = startForBlock; row < (startForBlock + blockSize); row++) {
        if (row >= rowsNumber) {
            break;
        }
        for (column = startForThread; column < (startForThread + threadSize); column++) {
            if (column >= columnsNumber) {
                break;
            }
            red = 0;
            green = 0;
            blue = 0;
            for (x = 0; x < 5; x++) //normalizacja
                for (y = 0; y < 5; y++) {
                    red += R[(row + x) * columnsNumber + column + y] * mask[x][y]; //nowy pixel (R)
                    green += G[(row + x) * columnsNumber + column + y] * mask[x][y]; //nowy pixel (G)
                    blue += B[(row + x) * columnsNumber + column + y] * mask[x][y]; //nowy pixel (B)
                }
            resultRed[row * columnsNumber + column] = red / SumMask; // dzielenie przez sume wszystkich wag maski
            resultGreen[row * columnsNumber + column] = green / SumMask;
            resultBlue[row * columnsNumber + column] = blue / SumMask;   
        }
    }
 
}
 
int main(int argc, char **argv)
{
    int blockNumber = 500; //cuda cores
    int threadNumber = 1024;
    int threadsCount; //liczba watkow
    hipEvent_t start, stop; //deklaracja zmiennych licznika
    float elapsedTime; 
    hipEventCreate(&start);
    hipEventCreate(&stop);

    if (argc != 4) {                //sprawdzenie ilosci argumentow podanych przy wywolaniu programu
        cout << "Niepoprawna liczba oargumentow"<<endl;
        exit(1);
    } 

    threadsCount = atoi(argv[1]);
    if (threadsCount <= 0 || threadsCount > 15) {
        cout << "Liczba rdzeni niepoprawna";
        exit(-1);
    }

    blockNumber=threadsCount;
    //Load and create image
    char *imgName = argv[2];
    char *imgOutName = argv[3];
    
    Mat img;
    img = imread(imgName, CV_LOAD_IMAGE_COLOR); // wczytanie obrazu wejsciowego
 
    if (!img.data) {
        cout << "Nie mozna wczytac" << imgName;
        getchar();
        return -1;
    }
 
    Mat background[3];   //destination array
    split(img, background); //split source
 
    //Prepare size of image and fragment sizes:
    int rows = img.rows;
    int columns = img.cols;
    int sizeForBlock = rows / blockNumber;
    int sizeForThread = columns / threadNumber;
   //cout << rows << "  " << columns << "  "<< sizeForBlock <<"  "<< sizeForThread  << endl;
 
    //Prepare data to upload
    unsigned char* R;
    unsigned char* G;
    unsigned char* B;
    unsigned char* cudaR;
    unsigned char* cudaG;
    unsigned char* cudaB;
 
    //Load data on GPU memory
    hipMalloc(&cudaR, rows*columns*sizeof(unsigned char));
    hipMalloc(&R, (rows)*(columns) * sizeof(unsigned char));
    hipMemcpy(cudaR, &background[2].data[0], rows*columns * sizeof(unsigned char),hipMemcpyHostToDevice);
    hipMalloc(&cudaG, rows*columns * sizeof(unsigned char));
    hipMalloc(&G, (rows)*(columns) * sizeof(unsigned char));
    hipMemcpy(cudaG, &background[1].data[0], rows*columns * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMalloc(&cudaB, rows*columns * sizeof(unsigned char));
    hipMalloc(&B, (rows)*(columns ) * sizeof(unsigned char));
    hipMemcpy(cudaB, &background[0].data[0], rows*columns * sizeof(unsigned char), hipMemcpyHostToDevice);
 
    //Do calculations
    hipEventRecord(start);
    gaussianBlur <<< blockNumber, threadNumber >>> (cudaR, cudaG, cudaB, R, G, B, sizeForBlock, sizeForThread, rows-4, columns-4); //uruchomienie jądra
    hipEventRecord(stop); //zatrzymanie licznika i zczytanie czasu obliczen
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Czas : %f ms\n", elapsedTime);
 
    //Wczytanie rozmytego obrazu z GPU
    unsigned char* resultBlue;
    unsigned char* resultRed;
    unsigned char* resultGreen;
 
    resultBlue = new unsigned char[(rows)*(columns)];
    resultRed = new unsigned char[(rows)*(columns)];
    resultGreen = new unsigned char[(rows)*(columns)];
 
    hipMemcpy(resultRed,R,(rows)*(columns) * sizeof(unsigned char),hipMemcpyDeviceToHost);
    hipMemcpy(resultBlue, B, (rows)*(columns) * sizeof(unsigned char), hipMemcpyDeviceToHost);
    hipMemcpy(resultGreen, G, (rows)*(columns) * sizeof(unsigned char), hipMemcpyDeviceToHost);
 
    Mat imgResult;
    Mat Red = Mat(rows, columns, CV_8UC1, resultRed);
    Mat Green = Mat(rows, columns, CV_8UC1, resultGreen);
    Mat Blue = Mat(rows, columns, CV_8UC1, resultBlue);
    vector<Mat> tab; //tablica do scalenia
 
    tab.push_back(Blue);
    tab.push_back(Green);
    tab.push_back(Red);
    merge(tab, imgResult); //scalanie 
    imwrite(imgOutName, Mat(imgResult, Rect(0, 0, columns - 4, rows - 4)));
 
    hipFree(&R);
    hipFree(&B);
    hipFree(&G);
    hipFree(&cudaR);
    hipFree(&cudaB);
    hipFree(&cudaG);
 

 
    return 0;
}
